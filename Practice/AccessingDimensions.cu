 #include<stdio.h>
 #include<hip/hip_runtime.h>
 #include<assert.h>
 

 __global__ void dkernel()
 {
  if(threadIdx.x==0 && blockIdx.x==0
  && threadIdx.y==0 && blockIdx.y==0
  && threadIdx.z==0 && blockIdx.z==0
  )
  {
    printf("%d %d %d %d %d %d \n",gridDim.x,gridDim.y,gridDim.z,
    blockDim.x,blockDim.y,blockDim.z);
  }
 }

 int main()
 {
  dim3 grid(2,3,4);
  dim3 block(5,6,7);
  dkernel<<<grid,block>>>();
  hipDeviceSynchronize();
  return 0;
 }