 #include<hip/hip_runtime.h>
 #include<stdio.h>
 #include<assert.h>
 

 __global__ void kernel(unsigned *matrix){
  unsigned id= threadIdx.x*blockDim.y+threadIdx.y;
  matrix[id]=id;
 }
 #define N 5
 #define M 6
 int main(){
  dim3 block(N,M,1);
  unsigned *matrix,*hmatrix;
  hipMalloc(&matrix,N*M*sizeof(unsigned));
  hmatrix=(unsigned*)malloc(N*M*sizeof(unsigned));
  kernel<<<1,block>>>(matrix);
  hipMemcpy(hmatrix,matrix,N*M*sizeof(unsigned),hipMemcpyDeviceToHost);
  for(unsigned i=0; i<N; i++){
    for(unsigned j=0; j<M; j++){
      printf("%d ",hmatrix[i*M+j]);
    }
    printf("\n");
  }
    return 0;
 }
