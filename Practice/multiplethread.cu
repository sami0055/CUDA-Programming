  #include<hip/hip_runtime.h>
 #include<stdio.h>
 
 #include<assert.h>
  __global__ void kernel(){
  printf("Hello world\n");
 }

 int main(){
  kernel<<<1,32>>>();
  hipDeviceSynchronize();
  return 0;
 }
 /*
   !nvcc multiplethread.cu -o multiplethread
   !./multiplethread
 */