 #include<hip/hip_runtime.h>
 #include<stdio.h>
 
 #include<assert.h>
 __global__ void kernel1(){
  printf("Hello world\n");
 }
 __global__ void kernel2(){
  printf("hello world2\n");
 }
 int main()
 {
  kernel1<<<1,1>>>();
  kernel2<<<1,1>>>();
  //printf("on CPU\n");
  hipDeviceSynchronize();
  printf("on CPU\n");

  return 0;
 }
 