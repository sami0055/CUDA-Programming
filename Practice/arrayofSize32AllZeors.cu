 #include <stdio.h>
#include<hip/hip_runtime.h>
#include<assert.h>

#define N 32
#define THREADS_PER_BLOCK 32

__global__ void initializeArray(int *arr) {
    int idx = threadIdx.x;
    
    if (idx < N) {
        arr[idx] = idx;
    }
}

int main() {
    int arr[N];
    int *d_arr;
    int size = N * sizeof(int);
    
    // Allocate memory on device
    hipMalloc(&d_arr, size);
    
    // Initialize array on device
    initializeArray<<<1,32>>>(d_arr);
    
    // Copy data back to host
    hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
    
    // Output the initialized array
    printf("Initialized array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
    
    // Free device memory
    hipFree(d_arr);
    
    return 0;
}
