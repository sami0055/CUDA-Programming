#include<stdio.h> 
#include<assert.h>
#include<hip/hip_runtime.h>


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}