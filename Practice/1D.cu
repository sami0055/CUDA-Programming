#include<stdio.h>
#include<assert.h>
#include<hip/hip_runtime.h>

__global__ void kernel(unsigned *matrix){
  unsigned id=blockIdx.x*blockDim.x+threadIdx.x;
  //blockDim=6 blockIdx= 0-4 threadIdx=0-5
  matrix[id]=id;
}
#define N 5
#define M 6

int main(){
unsigned *matrix,*hmatrix;
hipMalloc(&matrix,N*M*sizeof(unsigned));
hmatrix=(unsigned*)malloc(N*M*sizeof(unsigned));
kernel<<<N,M>>>(matrix);
hipMemcpy(hmatrix,matrix,N*M*sizeof(unsigned),hipMemcpyDeviceToHost);
for(unsigned i=0; i<N; i++)
{
  for(unsigned j=0; j<M; j++)
  {
    printf("%d ",hmatrix[i*M+j]);
  }
  printf("\n");
}
return 0;

}